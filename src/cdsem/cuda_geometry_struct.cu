#include "hip/hip_runtime.h"
/**
 * @file src/cdsem/cuda_geometry_struct.cu
 * @author Thomas Verduin <T.Verduin@tudelft.nl>
 * @author Sebastiaan Lokhorst <S.R.Lokhorst@tudelft.nl>
 */

#include "cuda_geometry_struct.cuh"
#include <algorithm>
#include <cfloat>
#include <functional>
#include <map>
#include <stack>
#include <vector>
#include <common/cuda_make_ptr.cuh>
#include <common/cuda_mem_scope.cuh>
#include <common/cuda_safe_call.cuh>

#include <iostream>

__host__ cuda_geometry_struct cuda_geometry_struct::create(const octree& root) {
    cuda_geometry_struct gstruct;
    gstruct.octree_dev_p = nullptr;
    gstruct.material_idx_in_dev_p = gstruct.material_idx_out_dev_p = nullptr;
    gstruct.triangle_Ax_dev_p = gstruct.triangle_Ay_dev_p = gstruct.triangle_Az_dev_p = nullptr;
    gstruct.triangle_Bx_dev_p = gstruct.triangle_By_dev_p = gstruct.triangle_Bz_dev_p = nullptr;
    gstruct.triangle_Cx_dev_p = gstruct.triangle_Cy_dev_p = gstruct.triangle_Cz_dev_p = nullptr;
    if(root.empty())
        return gstruct;

    // sort octree nodes by morton code.
    std::map<uint64_t,const octree*> morton_map;
    std::stack<const octree*> node_p_stack;
    node_p_stack.push(&root);
    while(!node_p_stack.empty()) {
        const octree* node_p = node_p_stack.top();
        node_p_stack.pop();
        morton_map[node_p->location()] = node_p;
        for(int octant = 0; octant < 8; octant++) {
            const octree* child_p = node_p->traverse(octant);
            if(child_p != nullptr)
                node_p_stack.push(child_p);
        }
    }

    // map octree nodes to indices.
    std::map<const octree*,int> node_p_map;
    for(auto cit = morton_map.cbegin(); cit != morton_map.cend(); cit++) {
        const int index = node_p_map.size();
        node_p_map[cit->second] = index;
    }

    // map triangles from octree to indices following morton code order.
    std::vector<const triangle*> triangle_p_vec;
    std::map<const triangle*,int> triangle_p_map;
    for(auto morton_cit = morton_map.cbegin(); morton_cit != morton_map.cend(); morton_cit++) {
        const octree* node_p = morton_cit->second;
        if(node_p->leaf())
            for(auto triangle_cit = node_p->cbegin(); triangle_cit != node_p->cend(); triangle_cit++)
                if(triangle_p_map.count(*triangle_cit) == 0) {
                    const int index = triangle_p_vec.size();
                    triangle_p_map[*triangle_cit] = index;
                    triangle_p_vec.push_back(*triangle_cit);
                }
    }

    // build linearized octree reference table
    //  i=0 : child does not exist
    //  i>0 : non-leaf child with node indices
    //  i<0 : leaf child with triangle indices (triangle index -1 means no triangle)
    const int occupancy = std::max(8, root.occupancy());
    std::vector<int> octree_vec(node_p_map.size()*occupancy);
    int index = 0;
    for(auto morton_cit = morton_map.cbegin(); morton_cit != morton_map.cend(); morton_cit++, index++) {
        const octree* node_p = morton_cit->second;
        if(node_p->leaf()) {
            int i = 0;
            for(auto triangle_cit = node_p->cbegin(); triangle_cit != node_p->cend(); triangle_cit++, i++)
                octree_vec[i+index*occupancy] = triangle_p_map[*triangle_cit];
            for(; i < occupancy; i++)
                octree_vec[i+index*occupancy] = -1;
        } else {
            for(int octant = 0; octant < 8; octant++) {
                const octree* child_p = node_p->traverse(octant);
                if(child_p != nullptr) {
                    int child_index = node_p_map[child_p];
                    if(child_p->leaf())
                        child_index = -child_index;
                    octree_vec[octant+index*occupancy] = child_index;
                }
            }
            int parent_index = 0;
            if(node_p->parent() != nullptr)
                parent_index = node_p_map[node_p->parent()];
            octree_vec[8+index*occupancy] = parent_index;
        }
    }

    // copy octree to device memory
    cuda_safe_call(__FILE__, __LINE__, [&]() {
        size_t pitch;
        hipMallocPitch(&gstruct.octree_dev_p, &pitch, occupancy*sizeof(int), node_p_map.size());
        cuda_mem_scope<int>(gstruct.octree_dev_p, pitch*node_p_map.size()/sizeof(int), [&](int* octree_p) {
            for(size_t index = 0; index < node_p_map.size(); index++)
            for(int i = 0; i < occupancy; i++)
                cuda_make_ptr<int>(octree_p, pitch, index)[i] = octree_vec[i+index*occupancy];
        });
        gstruct.octree_pitch = pitch;
        gstruct.root_center = make_float3(root.center().x, root.center().y, root.center().z);
        gstruct.root_size = make_float3(root.size().x, root.size().y, root.size().z);
        gstruct.occupancy = occupancy;
    });


    // copy triangles to device memory
    cuda_safe_call(__FILE__, __LINE__, [&]() {
        hipMalloc(&gstruct.material_idx_in_dev_p, triangle_p_vec.size()*sizeof(int));
        hipMalloc(&gstruct.material_idx_out_dev_p, triangle_p_vec.size()*sizeof(int));
        cuda_mem_scope<int>(gstruct.material_idx_in_dev_p, triangle_p_vec.size(), [&](int* in_p) {
        cuda_mem_scope<int>(gstruct.material_idx_out_dev_p, triangle_p_vec.size(), [&](int* out_p) {
            for(size_t i = 0; i < triangle_p_vec.size(); i++) {
                in_p[i] = triangle_p_vec[i]->in;
                out_p[i] = triangle_p_vec[i]->out;
            }
        });
        });
    });
    cuda_safe_call(__FILE__, __LINE__, [&]() {
        hipMalloc(&gstruct.triangle_Ax_dev_p, triangle_p_vec.size()*sizeof(float));
        hipMalloc(&gstruct.triangle_Ay_dev_p, triangle_p_vec.size()*sizeof(float));
        hipMalloc(&gstruct.triangle_Az_dev_p, triangle_p_vec.size()*sizeof(float));
        cuda_mem_scope<float>(gstruct.triangle_Ax_dev_p, triangle_p_vec.size(), [&](float* Ax_p) {
        cuda_mem_scope<float>(gstruct.triangle_Ay_dev_p, triangle_p_vec.size(), [&](float* Ay_p) {
        cuda_mem_scope<float>(gstruct.triangle_Az_dev_p, triangle_p_vec.size(), [&](float* Az_p) {
            for(size_t i = 0; i < triangle_p_vec.size(); i++) {
                Ax_p[i] = triangle_p_vec[i]->A.x;
                Ay_p[i] = triangle_p_vec[i]->A.y;
                Az_p[i] = triangle_p_vec[i]->A.z;
            }
        });
        });
        });
    });
    cuda_safe_call(__FILE__, __LINE__, [&]() {
        hipMalloc(&gstruct.triangle_Bx_dev_p, triangle_p_vec.size()*sizeof(float));
        hipMalloc(&gstruct.triangle_By_dev_p, triangle_p_vec.size()*sizeof(float));
        hipMalloc(&gstruct.triangle_Bz_dev_p, triangle_p_vec.size()*sizeof(float));
        cuda_mem_scope<float>(gstruct.triangle_Bx_dev_p, triangle_p_vec.size(), [&](float* Bx_p) {
        cuda_mem_scope<float>(gstruct.triangle_By_dev_p, triangle_p_vec.size(), [&](float* By_p) {
        cuda_mem_scope<float>(gstruct.triangle_Bz_dev_p, triangle_p_vec.size(), [&](float* Bz_p) {
            for(size_t i = 0; i < triangle_p_vec.size(); i++) {
                Bx_p[i] = triangle_p_vec[i]->B.x;
                By_p[i] = triangle_p_vec[i]->B.y;
                Bz_p[i] = triangle_p_vec[i]->B.z;
            }
        });
        });
        });
    });
    cuda_safe_call(__FILE__, __LINE__, [&]() {
        hipMalloc(&gstruct.triangle_Cx_dev_p, triangle_p_vec.size()*sizeof(float));
        hipMalloc(&gstruct.triangle_Cy_dev_p, triangle_p_vec.size()*sizeof(float));
        hipMalloc(&gstruct.triangle_Cz_dev_p, triangle_p_vec.size()*sizeof(float));
        cuda_mem_scope<float>(gstruct.triangle_Cx_dev_p, triangle_p_vec.size(), [&](float* Cx_p) {
        cuda_mem_scope<float>(gstruct.triangle_Cy_dev_p, triangle_p_vec.size(), [&](float* Cy_p) {
        cuda_mem_scope<float>(gstruct.triangle_Cz_dev_p, triangle_p_vec.size(), [&](float* Cz_p) {
            for(size_t i = 0; i < triangle_p_vec.size(); i++) {
                Cx_p[i] = triangle_p_vec[i]->C.x;
                Cy_p[i] = triangle_p_vec[i]->C.y;
                Cz_p[i] = triangle_p_vec[i]->C.z;
            }
        });
        });
        });
    });

    return gstruct;
}

__host__ void cuda_geometry_struct::release(cuda_geometry_struct& gstruct) {
    cuda_safe_call(__FILE__, __LINE__, [&]() {
        hipFree(gstruct.octree_dev_p);
    });
    gstruct.octree_dev_p = nullptr;
    cuda_safe_call(__FILE__, __LINE__, [&]() {
        for(int* p : {gstruct.material_idx_in_dev_p, gstruct.material_idx_out_dev_p})
            hipFree(p);
        for(float* p : {gstruct.triangle_Ax_dev_p, gstruct.triangle_Ay_dev_p, gstruct.triangle_Az_dev_p})
            hipFree(p);
        for(float* p : {gstruct.triangle_Bx_dev_p, gstruct.triangle_By_dev_p, gstruct.triangle_Bz_dev_p})
            hipFree(p);
        for(float* p : {gstruct.triangle_Cx_dev_p, gstruct.triangle_Cy_dev_p, gstruct.triangle_Cz_dev_p})
            hipFree(p);
    });
    gstruct.material_idx_in_dev_p = gstruct.material_idx_out_dev_p = nullptr;
    gstruct.triangle_Ax_dev_p = gstruct.triangle_Ay_dev_p = gstruct.triangle_Az_dev_p = nullptr;
    gstruct.triangle_Bx_dev_p = gstruct.triangle_By_dev_p = gstruct.triangle_Bz_dev_p = nullptr;
    gstruct.triangle_Cx_dev_p = gstruct.triangle_Cy_dev_p = gstruct.triangle_Cz_dev_p = nullptr;
}