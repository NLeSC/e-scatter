#include "hip/hip_runtime.h"
/**
 * @file src/cdsem/cuda_material_struct.cu
 * @author Thomas Verduin <T.Verduin@tudelft.nl>
 * @author Sebastiaan Lokhorst <S.R.Lokhorst@tudelft.nl>
 */

#include "cuda_material_struct.cuh"
#include <cuda_common/cuda_make_ptr.cuh>
#include <cuda_common/cuda_mem_scope.cuh>
#include <cuda_common/cuda_safe_call.cuh>
#include <common/constant.hh>

__host__ cuda_material_struct cuda_material_struct::create(int capacity) {
    cuda_material_struct mstruct;
    mstruct.capacity = capacity;
    cuda_safe_call(__FILE__, __LINE__, [&]() {
        size_t _pitch;
        hipMalloc(&mstruct.fermi_dev_p, capacity*sizeof(float));
        hipMalloc(&mstruct.barrier_dev_p, capacity*sizeof(float));
        hipMalloc(&mstruct.band_gap_dev_p, capacity*sizeof(float));
        hipMalloc(&mstruct.polaron_dev_p, capacity*sizeof(float));
        hipMalloc(&mstruct.phonon_loss_dev_p, capacity*sizeof(float));
        hipMallocPitch(&mstruct.elastic_dev_p, &_pitch, mstruct.K_cnt*sizeof(float), (mstruct.P_cnt+1)*capacity);
        hipMallocPitch(&mstruct.inelastic_dev_p, &_pitch, mstruct.K_cnt*sizeof(float), (mstruct.P_cnt+1)*capacity);
        hipMallocPitch(&mstruct.ionization_dev_p, &_pitch, mstruct.K_cnt*sizeof(float), (mstruct.P_cnt+1)*capacity);
        mstruct.pitch = _pitch;
    });
    return mstruct;
};

__host__ void cuda_material_struct::release(cuda_material_struct& mstruct) {
    cuda_safe_call(__FILE__, __LINE__, [&]() {
        hipFree(mstruct.fermi_dev_p);
        hipFree(mstruct.barrier_dev_p);
        hipFree(mstruct.band_gap_dev_p);
        hipFree(mstruct.polaron_dev_p);
        hipFree(mstruct.phonon_loss_dev_p);
        hipFree(mstruct.elastic_dev_p);
        hipFree(mstruct.inelastic_dev_p);
        hipFree(mstruct.ionization_dev_p);
    });
}

__host__ void cuda_material_struct::assign(int i, const material& _material) {
    if((i < 0) || (i >= capacity))
        return;
    cuda_safe_call(__FILE__, __LINE__, [&]() {
        cuda_mem_scope<float>(fermi_dev_p, capacity, [&](float* fermi_p) {
            fermi_p[i] = _material.fermi()/constant::ec;
        });
        cuda_mem_scope<float>(barrier_dev_p, capacity, [&](float* barrier_p) {
            barrier_p[i] = _material.barrier()/constant::ec;
        });
        cuda_mem_scope<float>(band_gap_dev_p, capacity, [&](float* band_gap_p) {
            if(_material.band_gap().is_defined())
                band_gap_p[i] = _material.band_gap()()/constant::ec;
            else
                band_gap_p[i] = -1;
        });
        cuda_mem_scope<float>(polaron_dev_p, capacity, [&](float* polaron_p) {
            polaron_p[i] = 0.0f;
            #warning "polaron parameter is determined for pmma by using string comparison"
            if(_material.name() == "pmma" || _material.name() == "pmma-surface")
                polaron_p[i] = 1.0f;
        });
        cuda_mem_scope<float>(phonon_loss_dev_p, capacity, [&](float* phonon_loss_p) {
            phonon_loss_p[i] = _material.phonon_loss()/constant::ec;
            #warning "phonon loss set to zero for pmma by using string comparison"
            if(_material.name() == "pmma" || _material.name() == "pmma-surface")
                phonon_loss_p[i] = 0.0f;
        });
    });
    auto __logspace_K_at = [&](int x) {
        return K_min*std::exp(1.0*x/(K_cnt-1)*std::log(K_max/K_min));
    };
    auto __linspace_P_at = [&](int y) {
        return 1.0*y/(P_cnt-1);
    };
    cuda_safe_call(__FILE__, __LINE__, [&]() {
        float* elastic_imfp_dev_p = cuda_make_ptr<float>(elastic_dev_p, pitch, P_cnt+1, 0, i);
        float* inelastic_imfp_dev_p = cuda_make_ptr<float>(inelastic_dev_p, pitch, P_cnt+1, 0, i);
        cuda_mem_scope<float>(elastic_imfp_dev_p, K_cnt, [&](float* elastic_imfp_p) {
        cuda_mem_scope<float>(inelastic_imfp_dev_p, K_cnt, [&](float* inelastic_imfp_p) {
            for(int x = 0; x < K_cnt; x++) {
                const double K = __logspace_K_at(x)*constant::ec;
                elastic_imfp_p[x] = std::log(_material.density()*_material.elastic_tcs(K)*1e-9);
                inelastic_imfp_p[x] = std::log(_material.density()*_material.inelastic_tcs(K)*1e-9);
            }
        });
        });
    });
    cuda_safe_call(__FILE__, __LINE__, [&]() {
        float* elastic_icdf_dev_p = cuda_make_ptr<float>(elastic_dev_p, pitch, P_cnt+1, 1, i);
        float* inelastic_icdf_dev_p = cuda_make_ptr<float>(inelastic_dev_p, pitch, P_cnt+1, 1, i);
        cuda_mem_scope<float>(elastic_icdf_dev_p, pitch, make_int2(K_cnt, P_cnt), [&](float** elastic_icdf_p) {
        cuda_mem_scope<float>(inelastic_icdf_dev_p, pitch, make_int2(K_cnt, P_cnt), [&](float** inelastic_icdf_p) {
            for(int y = 0; y < P_cnt; y++) {
                const double P =  __linspace_P_at(y);
                for(int x = 0; x < K_cnt; x++) {
                    const double K = __logspace_K_at(x)*constant::ec;
                    elastic_icdf_p[y][x] = std::cos(_material.elastic_icdf(K, P));
                    inelastic_icdf_p[y][x] = std::log(_material.inelastic_icdf(K, P)/constant::ec);
                }
            }
        });
        });
    });
    cuda_safe_call(__FILE__, __LINE__, [&]() {
        float* binding_dev_p = cuda_make_ptr<float>(ionization_dev_p, pitch, P_cnt+1, 0, i);
        cuda_mem_scope<float>(binding_dev_p, pitch, make_int2(K_cnt, P_cnt), [&](float** binding_p) {
            for(int y = 0; y < P_cnt; y++) {
                const double P = __linspace_P_at(y);
                for(int x = 0; x < K_cnt; x++) {
                    const double omega0 = __logspace_K_at(x);
                    const double margin = 0; // magic number in accordance with Kieft & Bosch code (old value = 10)
                    double binding = -1;
                    if(omega0 > 100) {
                        binding = _material.ionization_energy((omega0+margin)*constant::ec, P)/constant::ec;
                        if(binding < 50)
                            binding = -1;
                    }
                    if(binding < 0) {
                        binding = _material.outer_shell_ionization_energy(omega0*constant::ec)/constant::ec;
                        if(binding < 0)
                            binding = -1;
                    }
                    binding_p[y][x] = binding;
                }
            }
        });
    });
}
